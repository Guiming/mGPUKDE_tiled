// Copyright 2022 Guiming Zhang (guiming.zhang@du.edu)
// Distributed under GNU General Public License (GPL) license

#include "CUDA_KDtree.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <cstdio>

void CheckCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

CUDA_KDTree::~CUDA_KDTree()
{
    hipFree(m_gpu_nodes);
    hipFree(m_gpu_indexes);
    hipFree(m_gpu_points);
}

void CUDA_KDTree::CreateKDTree(KDNode *root, int num_nodes, const vector <Point> &data)
{
    // Create the nodes again on the CPU, laid out nicely for the GPU transfer
    // Not exactly memory efficient, since we're creating the entire tree again
    m_num_points = data.size();

    for (int i = 0; i < 2; i++)
    {
        hipMalloc((void**)&m_gpu_nodes, sizeof(CUDA_KDNode) * num_nodes);
        hipMalloc((void**)&m_gpu_indexes, sizeof(int) * m_num_points);
        hipMalloc((void**)&m_gpu_points, sizeof(Point) * m_num_points);
    }
    CheckCUDAError("CreateKDTree");

    vector <CUDA_KDNode> cpu_nodes(num_nodes);
    vector <int> indexes(m_num_points);
    vector <KDNode*> to_visit;

    int cur_pos = 0;

    to_visit.push_back(root);
    int counter = 0;
    while(to_visit.size()) {
        
        

        vector <KDNode*> next_search;
        
        int counter2 = 0;

        while(to_visit.size()) {

            //printf("***still building KDTree (GPU) - %d -> %d\n", counter, counter2);

            KDNode *cur = to_visit.back();
            to_visit.pop_back();

            int id = cur->id;

            cpu_nodes[id].level = cur->level;
            cpu_nodes[id].parent = cur->_parent;
            cpu_nodes[id].left = cur->_left;
            cpu_nodes[id].right = cur->_right;
            cpu_nodes[id].split_value = cur->split_value;
            cpu_nodes[id].num_indexes = cur->indexes.size();

            if(cur->indexes.size()) {
                for(unsigned int i=0; i < cur->indexes.size(); i++)
                    indexes[cur_pos+i] = cur->indexes[i];

                cpu_nodes[id].indexes = cur_pos;
                cur_pos += cur->indexes.size();
            }
            else {
                cpu_nodes[id].indexes = -1;
            }

            if(cur->left)
                next_search.push_back(cur->left);

            if(cur->right)
                next_search.push_back(cur->right);
            
            counter2 += 1;
        }

        to_visit = next_search;

        counter += 1;
    }

    hipMemcpy(m_gpu_nodes, &cpu_nodes[0], sizeof(CUDA_KDNode)*cpu_nodes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_indexes, &indexes[0], sizeof(int)*indexes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_points, &data[0], sizeof(Point)*data.size(), hipMemcpyHostToDevice);

    CheckCUDAError("CreateKDTree");
}

